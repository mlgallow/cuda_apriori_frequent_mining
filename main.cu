#include "hip/hip_runtime.h"
//#include <stdio.h>
#include <iostream>
#include<algorithm>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>
#include <fstream>
#include "defs.h"
#include "support.h"
#include "kernel.cu"
#include<vector>
#include<utility>
using namespace std;


bool pair_compare(const pair<short unsigned int, unsigned int>& p1,const pair<short unsigned int, unsigned int>& p2);
int main(int argc, char* argv[])
{
    char *line = NULL;
    size_t len = 0;
    unsigned int lines = 0;
    unsigned int count = 0;
    char *ln, *nptr;

    unsigned int *transactions = NULL;
    unsigned int *trans_offset = NULL;
    unsigned int *ci_h = NULL;//bins array for histogram op
    //unsigned int *flist = NULL;
    //unsigned short *flist_key_16_index = NULL;

    unsigned int element_id = 0;

    transactions = (unsigned int *) malloc(MAX_NUM_ELEMENTS * sizeof(unsigned int));
    trans_offset = (unsigned int *) malloc((MAX_NUM_ELEMENTS + 1) * sizeof(unsigned int));
    ci_h = (unsigned int *) malloc(MAX_UNIQUE_ITEMS * sizeof(unsigned int));
    //flist = (unsigned int *) malloc(max_unique_items * sizeof(unsigned int));
    //flist_key_16_index = (unsigned short*) malloc(max_unique_items * sizeof(unsigned short));

//    memset(flist_key_16_index, 0xFFFF, max_unique_items * sizeof(unsigned short));

    lines = 0;
    element_id = 0;
    ifstream fp1("topic-3.txt");
    string curline, space(" ");
    if (fp1.is_open()) {
        cout<<"file opened"<<endl;
        trans_offset[0] = 0;
        while(getline(fp1, curline) && lines < MAX_TRANSACTIONS) {
            count = 0;
            istringstream s(curline); 
            string st;
            while(getline(s, st, ' ') && count < MAX_ITEM_PER_TRANSACTION) {
                int item = atol(st.c_str());
                if (item < MAX_UNIQUE_ITEMS) {
                    // add an item only if it is in the range [0,max_unique_items)
                    transactions[element_id++] = atol(st.c_str());
                    count++;
                }
            }
            if (count > 0) {
                // consider this transaction if there is atleast one item
                trans_offset[lines + 1] = trans_offset[lines] + count;
                lines++;
            }
        }
    } else {
        cout<<"error in reading from file"<<endl;
        return 0;
    }
    fp1.close();
    unsigned int num_elements = element_id;
    unsigned int num_transactions = lines;
    cout<<"Number of Transactions = "<<num_transactions<<endl;
    cout<<"num_elements in transactions array = "<<num_elements<<endl;
    #ifdef TEST_PARAMS
    for (int i = 0; i < num_elements; i++){
        cout<<transactions[i]<<" ";
    }
    cout<<endl;
    for (int i = 0; i <= num_transactions; i++) {
       cout<<"(i,offset)"<<i<<","<<trans_offset[i]; 
    }
    #endif

    //calculate max power
    int power = 1;
    while ((MAX_UNIQUE_ITEMS / (int)(pow(10.0, (double)power))) != 0) {
        power += 1;
    }
    cout<<"max power = "<<power<<endl;

    //check for max item exceed
    if (num_elements > MAX_NUM_ELEMENTS) {
        cout<<"Error: Elements exceeding NUM_ELEMENTS. Exiting...";
        return -1;
    }
    /////////////////////////////////////////////////////////////////////////////////////
    /////////////////////// Device Variables Initializations ///////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////
    unsigned int *d_input;//
    unsigned int *d_offsets;
    unsigned int *ci_d;//bins array - each index corrosponds to an item
    hipDeviceProp_t deviceProp;
    Timer timer;
    hipError_t cuda_ret;
    hipGetDeviceProperties(&deviceProp, 0);
    cout<<"Allocating device variables...";
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&d_input, num_elements * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&d_offsets, (num_transactions + 1) * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    cuda_ret = hipMalloc((void**)&ci_d, MAX_UNIQUE_ITEMS * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMemset(ci_d, 0, MAX_UNIQUE_ITEMS * sizeof(unsigned int));
    
    cuda_ret = hipMemcpy(d_input, transactions, num_elements * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy input to the device");
    dim3 grid_dim, block_dim;
    block_dim.x = BLOCK_SIZE; 
    block_dim.y = 1; block_dim.z = 1;
    grid_dim.x = ceil(num_elements / (1.0 * BLOCK_SIZE)); 
    grid_dim.y = 1; grid_dim.z = 1;
    cout<<"launching histogram kernel(grid, block):"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    histogram_kernel<<<grid_dim, block_dim, MAX_UNIQUE_ITEMS * sizeof(unsigned int)>>>(d_input, ci_d, num_elements);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch Histogram kernel");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    hipMemcpy(ci_h, ci_d, MAX_UNIQUE_ITEMS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    cout<<"histogram output:"<<endl;
    for (int i = 0; i < MAX_UNIQUE_ITEMS; i++) {
        cout<<"ci_h["<<i<<"]="<<ci_h[i]<<endl;   
    }     
#ifdef TEST_PARAMS
#endif
#if 0

    unsigned int *d_flist, *d_flist_key_16;
    unsigned short *d_flist_key_16_index;
    int SM_PER_BLOCK = deviceProp.sharedMemPerBlock;
    int CONST_MEM_GPU = deviceProp.totalConstMem;
    // Allocate device variables ----------------------------------------------


    cuda_ret = hipMalloc((void**)&d_flist, max_unique_items * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&d_flist_key_16_index, max_unique_items * sizeof(unsigned short));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&d_flist_key_16, max_unique_items * sizeof(unsigned short));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");


    hipDeviceSynchronize();
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    cuda_ret = hipMemcpy(d_transactions, transactions, num_items_in_transactions * sizeof(unsigned int),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");
    // to test
	cuda_ret = hipMemcpy(d_trans_offsets, trans_offset, (num_transactions + 1) * sizeof(unsigned int),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");
    
    cuda_ret = hipMemset(d_flist, 0, max_unique_items * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to set device memory");
    
    startTime(&timer);
    cout<<"histogram kernel\n";
    make_flist(d_trans_offsets, d_transactions, d_flist, num_transactions, num_items_in_transactions, SM_PER_BLOCK);
    // now prune flist
    dim3 block_dim;
    dim3 grid_dim;
    
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;

    grid_dim.x = (int) ceil(max_unique_items / (1.0 * block_dim.x));
    grid_dim.y = 1;
    grid_dim.z = 1;

    pruneList<<<grid_dim, block_dim>>>(d_flist, max_unique_items, support);
    cout<<"constant mem available:"<<CONST_MEM_GPU<<endl;
    startTime(&timer);
    cout<<"copying flist form dev to host\n";
    cuda_ret = hipMemcpy(flist, d_flist, max_unique_items * sizeof(unsigned int),
        hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
    hipDeviceSynchronize();
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;

    // now keep track of only actual number of items which have passed support count
    int *li_h = (int *) malloc(max_unique_items * sizeof(int));
    int actualNumItems = 0;
    for (int i =0; i < max_unique_items;i++) {
        if (flist[i] != 0) {
            li_h[actualNumItems++] = i;
        } 
    }
    
    int*li_d;
    cuda_ret = hipMalloc((void**)&li_d, actualNumItems * sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    int maskLength = pow(float(actualNumItems), 2);
    cout <<"atual item ="<<actualNumItems;
    cout <<"maskLength ="<<maskLength<<endl;
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;

    grid_dim.x = (int) ceil((maskLength) / (1.0 * block_dim.x));
    grid_dim.y = 1;
    grid_dim.z = 1;
    int *mask_h = (int*)malloc(maskLength * sizeof(int));
    int* d_mask;
    cout<<"alloc mask"<<endl;
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&d_mask, maskLength * sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    
    cout<<"init mask"<<endl;
    startTime(&timer);
    initializeMaskArray<<<grid_dim, block_dim>>>(d_mask, maskLength);
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    hipDeviceSynchronize();

    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil((maskLength) / (1.0 * MAX_ITEM_PER_SM));
    grid_dim.y = 1;
    grid_dim.z = 1;
    cout<<"self join launched with <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    selfJoinKernel<<<grid_dim, block_dim>>>(li_d, d_mask, actualNumItems);
    hipDeviceSynchronize();
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    hipDeviceSynchronize();
    // Free memory ------------------------------------------------------------

    #ifdef TEST_MODE
    int*h_mask;
    h_mask = (int*)malloc(maskLength * sizeof(int));
    hipMemcpy(h_mask, d_mask, maskLength * sizeof(int),
       hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    for (int i =0;i < maskLength;i++) {
        cout<<h_mask[i]<<",";    
    }
        /*for (int i =0;i < actualNumItems;i++) {
            cout<<li_h[i]<<",";
        }*/
    free(h_mask);
    #endif
    hipFree(d_flist);
    hipFree(d_flist_key_16);
    hipFree(d_flist_key_16_index);
    hipFree(d_mask);
    
    free(li_h);
    free(flist);
    //free(flist_key_16);
    free(flist_key_16_index);
#endif
    free(trans_offset);
    free(transactions);
    free(ci_h);
    hipFree(d_offsets);
    hipFree(d_input);
    hipFree(ci_d);
    cout<<"program end";

}

bool pair_compare(const pair<short unsigned int, unsigned int>& p1,const pair<short unsigned int, unsigned int>& p2) {
    return p1.second < p2.second;    
}
