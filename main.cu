#include "hip/hip_runtime.h"
//#include <stdio.h>
#include <iostream>
#include<algorithm>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>
#include <fstream>
#include "defs.h"
#include "support.h"
#include "kernel.cu"
#include "kernel_prescan.cu"
#include<vector>
#include<map>
#include<utility>
#include<algorithm>
using namespace std;

class tuple {
    // ensure it doesnt exceed MAX_PATTERN
    public:
    vector<int> values;
    tuple(int val1) {
        values.push_back(val1);
    }
    tuple() {}
    tuple(int val1, int val2) {
        values.push_back(val1);
        values.push_back(val2);
        std::sort(values.begin(), values.end());    
    }
    tuple(int val1, int val2, int val3) {
        values.push_back(val1);
        values.push_back(val2);
        values.push_back(val3);
        std::sort(values.begin(), values.end());    
    }
   
    int get(int index) {
        if (index >= values.size()) index = values.size();
        else if (index < 0) index = 0;
        
        return values[index];  
    } 
    
    int size() {
       return values.size(); 
    }
    
    void print() {
        vector<int>::iterator it = values.begin();
        cout<<"(";
        while(it != values.end()) {
            cout<<*it<<",";
            it++; 
        }
        cout<<")";
    }
    bool operator==(const tuple &other) const {
        if (values.size() != (other.values).size()) return false;
        vector<int>::const_iterator it, it_other;
        for (it = values.begin(), it_other = other.values.begin(); it != values.end();it++, it_other++) {
            if (*it != *it_other) return false;
        }
        return true;
    }
    
    bool insertValues(int item) {
        if (values.size() >= MAX_PATTERN_SEARCH) return false;
        values.push_back(item);
        return true;
    }
    
    tuple getFirstNitems(int n) {
        if (n > values.size()) n = values.size();
        vector<int>::iterator it = values.begin();
        tuple op;
        for (int i = 0;i < n;i++) {
            op.insertValues(*it++);
        }
        return op;
    }

    tuple getLastItem() {
        return values[values.size() - 1];
    }
};

bool isTuplePresent(const vector<std::pair<tuple, int> >&list, const tuple &t) {
    if (list.size() == 0) return false;
    vector<std::pair<tuple, int> >::const_iterator it = list.begin();
    while(it != list.end()) {
        tuple cur_tuple = it->first;
        if (cur_tuple == t) return true;
        it++;
    }
    return false;
}

int  getTupleValue(const vector<std::pair<tuple, int> >&list, const tuple &t) {
    if (list.size() == 0) return INVALID;
    vector<std::pair<tuple, int> >::const_iterator it = list.begin();
    while(it != list.end()) {
        tuple cur_tuple = it->first;
        if (cur_tuple == t) {
            return it->second;  
        }
        it++;
    }
    return false;
}

int compare(const void *a, const void *b) {
    int a1 = *((int*)a);
    int b1 = *((int*)b);
    if (a1 == b1) return 0;
    else if (a1 < b1) return -1;
    else return 1;
}
bool pair_compare(const pair<short unsigned int, unsigned int>& p1,const pair<short unsigned int, unsigned int>& p2);
int main(int argc, char* argv[])
{
    float totalRunTime = 0.0;
    float totalAllocTime = 0.0;
    char *line = NULL;
    size_t len = 0;
    unsigned int lines = 0;
    unsigned int count = 0;
    char *ln, *nptr;

    unsigned int *transactions = NULL;
    unsigned int *trans_offset = NULL;
    unsigned int *ci_h = NULL;//bins array for histogram op

    unsigned int element_id = 0;

    transactions = (unsigned int *) malloc(MAX_NUM_ELEMENTS * sizeof(unsigned int));
    trans_offset = (unsigned int *) malloc((MAX_NUM_ELEMENTS + 1) * sizeof(unsigned int));
    ci_h = (unsigned int *) malloc(MAX_UNIQUE_ITEMS * sizeof(unsigned int));

    lines = 0;
    element_id = 0;
    ifstream fp1("topic-3.txt");
    string curline, space(" ");
    if (fp1.is_open()) {
        cout<<"file opened"<<endl;
        trans_offset[0] = 0;
        while(getline(fp1, curline) && lines < MAX_TRANSACTIONS) {
            count = 0;
            istringstream s(curline); 
            string st;
            while(getline(s, st, ' ') && count < MAX_ITEM_PER_TRANSACTION) {
                int item = atol(st.c_str());
                if (item < MAX_UNIQUE_ITEMS) {
                    // add an item only if it is in the range [0,max_unique_items)
                    transactions[element_id++] = atol(st.c_str());
                    count++;
                }
            }
            if (count > 0) {
                // consider this transaction if there is atleast one item
                trans_offset[lines + 1] = trans_offset[lines] + count;
                lines++;
            }
        }
    } else {
        cout<<"error in reading from file"<<endl;
        return 0;
    }
    fp1.close();
    unsigned int num_elements = element_id;
    unsigned int num_transactions = lines;
    cout<<"Number of Transactions = "<<num_transactions<<endl;
    cout<<"num_elements in transactions array = "<<num_elements<<endl;
    #ifdef TEST_PARAMS
    /*for (int i = 0; i < num_elements; i++){
        cout<<transactions[i]<<" ";
    }
    cout<<endl;
    for (int i = 0; i <= num_transactions; i++) {
       cout<<"(i,offset)"<<i<<","<<trans_offset[i]; 
    }*/
    #endif

    //calculate max power
    int power = 1;
    while ((MAX_UNIQUE_ITEMS / (int)(pow(10.0, (double)power))) != 0) {
        power += 1;
    }
    cout<<"max power = "<<power<<endl;

    //check for max item exceed
    if (num_elements > MAX_NUM_ELEMENTS) {
        cout<<"Error: Elements exceeding NUM_ELEMENTS. Exiting...";
        return -1;
    }
    /////////////////////////////////////////////////////////////////////////////////////
    /////////////////////// Device Variables Initializations ///////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////
    unsigned int *d_input;//
    unsigned int *d_offsets;
    unsigned int *ci_d;//bins array - each index corrosponds to an item
    hipDeviceProp_t deviceProp;
    Timer timer;
    hipError_t cuda_ret;
    hipGetDeviceProperties(&deviceProp, 0);
    cout<<"Allocating device variables...";
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&d_input, num_elements * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&d_offsets, (num_transactions + 1) * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&ci_d, MAX_UNIQUE_ITEMS * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMemset(ci_d, 0, MAX_UNIQUE_ITEMS * sizeof(unsigned int));
    cuda_ret = hipMemcpy(d_input, transactions, num_elements * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy input to the device");
    cuda_ret = hipMemcpy(d_offsets, trans_offset, (num_transactions+1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy input to the device");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
    
    
    dim3 grid_dim, block_dim;
    block_dim.x = BLOCK_SIZE; 
    block_dim.y = 1; block_dim.z = 1;
    grid_dim.x = ceil(num_elements / (1.0 * BLOCK_SIZE)); 
    grid_dim.y = 1; grid_dim.z = 1;
    //################################################################################//
    //###########################histogram_kernel####################################//
    cout<<"launching histogram kernel(grid, block):"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    histogram_kernel<<<grid_dim, block_dim, MAX_UNIQUE_ITEMS * sizeof(unsigned int)>>>(d_input, ci_d, num_elements);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch Histogram kernel");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    // prune the histogram op 
    block_dim.x = BLOCK_SIZE; 
    block_dim.y = 1; block_dim.z = 1;
    grid_dim.x = ceil(MAX_UNIQUE_ITEMS / (1.0 * BLOCK_SIZE)); 
    grid_dim.y = 1; grid_dim.z = 1;
    //################################################################################//
    //###########################pruneGPU_kernel####################################//
    cout<<"launching pruning kernel(grid, block):"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    pruneGPU_kernel<<<grid_dim, block_dim>>>(ci_d, MAX_UNIQUE_ITEMS, MIN_SUPPORT);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
    
    cout<<"copying hist op back to host"<<endl;
    startTime(&timer);
    hipMemcpy(ci_h, ci_d, MAX_UNIQUE_ITEMS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
#ifdef TEST_PARAMS
    cout<<"histogram output after pruning:"<<endl;
    for (int i = 0; i < MAX_UNIQUE_ITEMS; i++) {
        cout<<"ci_h["<<i<<"]="<<ci_h[i]<<endl;   
    }   
#endif
    startTime(&timer);
    unsigned int *li_h; // this list contains the actual items which passed min support test
    unsigned int  k = 0; //count of actual items which passed min support test
    for (int i = 0;i<MAX_UNIQUE_ITEMS;i++) {
        if (ci_h[i] != 0) {
            k++;    
        }    
    }
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
    cout<<"num items with good support count="<<k<<endl;
    startTime(&timer);
    li_h = (unsigned int *) malloc(k * sizeof(unsigned int));
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
    
    
    startTime(&timer);
    int li_count = 0;
    for (int i = 0;i<MAX_UNIQUE_ITEMS;i++) {
        if (ci_h[i] != 0) {
            li_h[li_count++] = i; 
        } 
    }
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);

#ifdef TEST_PARAMS
    cout<<"li_h after pruning:"<<endl;
    for (int i = 0; i < k; i++) {
        cout<<"li_h["<<i<<"]="<<li_h[i]<<endl;   
    }
#endif
    unsigned int *li_d;
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&li_d, k * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemcpy(li_d, li_h, k * sizeof(unsigned int), hipMemcpyHostToDevice);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy li_h to device");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
    
    int maskLength = pow(float(k), 2);
    cout <<"maskLength ="<<maskLength<<endl;
    int *mask_h = (int*)malloc(maskLength * sizeof(int));
    int* mask_d;//mask matrix
    cout<<"alloc mask matrix"<<endl;
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&mask_d, maskLength * sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
    
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil((maskLength) / (1.0 * block_dim.x));
    grid_dim.y = 1;
    grid_dim.z = 1;
    cout<<"init mask"<<endl;
    startTime(&timer);
    initializeMaskArray<<<grid_dim, block_dim>>>(mask_d, maskLength);
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    hipDeviceSynchronize();
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * MAX_ITEM_PER_SM));
    grid_dim.y = 1;
    grid_dim.z = 1;
    //################################################################################/
    //###########################selfJoinKernel#######################################/
    cout<<"self join launched with <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    selfJoinKernel<<<grid_dim, block_dim>>>(li_d, mask_d, k, power);
    hipDeviceSynchronize();
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
  
    // TBD:to test. remove in final code
    cout<<"copy mask back to host"<<endl;
    startTime(&timer);
    hipMemcpy(mask_h, mask_d, maskLength * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
#ifdef TEST_PARAMS
    /*cout<<"################mask_h after join#############"<<endl;
    for (int i = 0;i < maskLength; i++) {
        cout<<"mask["<<i<<"]="<<mask_h[i]<<endl;   
        
    }*/
#endif
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil((num_transactions) / (1.0 * MAX_TRANSACTION_PER_SM));
    grid_dim.y = 1;
    grid_dim.z = 1;
    //################################################################################/
    //###########################findFrequencyGPU#######################################/
    cout<<"findFrequencyGPU <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    findFrequencyGPU_kernel<<<grid_dim, block_dim>>>(d_input, d_offsets, num_transactions, num_elements, li_d, mask_d, k, maskLength);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch findFrequencyGPU_kernel");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    //prune the 2d mask matrix
    block_dim.x = BLOCK_SIZE;
    block_dim.y = BLOCK_SIZE;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = (int) ceil(k / (1.0 * block_dim.y));
    grid_dim.z = 1;
    //################################################################################/
    //###########################pruneMultipleGPU_kernel##############################
    cout<<"pruneMultipleGPU <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    pruneMultipleGPU_kernel<<<grid_dim, block_dim>>>(mask_d, k, MIN_SUPPORT);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch pruneMultipleGPU");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);

    cout<<"copy mask back to host"<<endl;
    startTime(&timer);
    hipMemcpy(mask_h, mask_d, maskLength * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
#ifdef TEST_PARAMS
    cout<<"################mask_h after findFrequencyGPU_kernel and Prune#############"<<endl;
    for (int i = 0;i < maskLength; i++) {
        cout<<"mask["<<i<<"]="<<mask_h[i]<<endl;   
        
    }
#endif
    //now we need to convert the mask array to a sparse matrix in parallel
    // this means we need to find number of non zero entries in each row of mask matrix
    // and the allocate memory equal to total number of non zero items.
    // each thread can then directly work on an offset into the array, 
    // obtained by perorming a exclusive scan.
    unsigned int *ci_dn;
    unsigned int *ci_hn;
    ci_hn = (unsigned int*) malloc(k * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&ci_dn, k * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(ci_dn, 0, k * sizeof(unsigned int));
    
    block_dim.x = BLOCK_SIZE;
    block_dim.y = BLOCK_SIZE;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = (int) ceil(k / (1.0 * block_dim.y));
    grid_dim.z = 1;

    //################################################################################/
    //###########################combinationsAvailable_kernel##############################
    cout<<"combinationsAvailable_kernel <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    combinationsAvailable_kernel<<<grid_dim, block_dim>>>(mask_d, ci_dn, k, maskLength);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch combinationsAvailable_kernel");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
    cout<<"Copy ci_dn to host"<<endl;
    startTime(&timer);
    hipMemcpy(ci_hn, ci_dn, k * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
#ifdef TEST_PARAMS
    for (int i = 0; i < k; i++) {
        cout<<"ci_dn["<<i<<"]="<<ci_hn[i]<<endl;    
    }
#endif
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    // prescan the ci_hn array
    unsigned int *ci_hnx;
    unsigned int *ci_dnx;
    startTime(&timer);
    ci_hnx = (unsigned int*) malloc(k * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&ci_dnx, k * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(ci_dnx, 0, k * sizeof(unsigned int));
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
    
    //###############################################################
    //###########################preScan##############################
    cout<<"prescan time"<<endl; 
    startTime(&timer);
    preScan(ci_dnx, ci_dn, k);
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
    
    hipMemcpy(ci_hnx, ci_dnx, k * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
#ifdef TEST_PARAMS
    cout<<"scan op"<<endl;
    for (int i = 0; i < k; i++) {
        cout<<"ci_dnx["<<i<<"]="<<ci_hnx[i]<<endl;    
    }
#endif

    unsigned int *sparseM_h;
    unsigned int *sparseM_d;
    unsigned int sparse_matrix_size = ci_hnx[k-1];
    cout<<"allocating sparse matrix for size"<<sparse_matrix_size<<endl; 
    sparseM_h = (unsigned int*) malloc(3 * sparse_matrix_size * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&sparseM_d, 3 * sparse_matrix_size * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(sparseM_d, 0, 3 * sparse_matrix_size * sizeof(unsigned int));
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = 1;
    grid_dim.z = 1;
    cout<<" convert2Sparse kernel <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    convert2Sparse<<<grid_dim, block_dim>>>(mask_d, ci_dnx, sparseM_d, sparse_matrix_size, k);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch convert2Sparse");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
    startTime(&timer);
    hipMemcpy(sparseM_h, sparseM_d, 3 * sparse_matrix_size * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
#ifdef TEST_PARAMS
    cout<<"sparse op(row,col,val)"<<endl;
    for (int i = 0; i < sparse_matrix_size; i++) {
        cout<<"sparse("<<sparseM_h[i]<<","<<sparseM_h[i + sparse_matrix_size]<<","<<sparseM_h[i + 2*sparse_matrix_size]<<")"<<endl;    
    }
#endif
    //now create a STL map and add the sparse matrix values to the map
    cout<<"Making vector modulo map"<<endl;
    startTime(&timer);
    vector<std::pair<tuple, int> > patterns;
    //cout<<"build vector from sparse array of length = "<<sparse_matrix_size<<endl;
    for (int i = 0; i< sparse_matrix_size;i++) {
        tuple t(li_h[sparseM_h[i]], li_h[sparseM_h[i + sparse_matrix_size]]);
        int item = sparseM_h[i + 2 * sparse_matrix_size];
        patterns.push_back(std::pair<tuple, unsigned int>(t, item));    
    }
    //cout<<"map size"<<patterns.size()<<endl;
#ifdef TEST_PARAMS
    vector<std::pair<tuple, int> >::iterator it;
    for (it = patterns.begin(); it != patterns.end();it++) {
        it->first.print();
        cout<<"="<<it->second<<endl;    
    }
#endif
    vector<std::pair<tuple, int> > new_modulo_map;
    //cout<<isTuplePresent(dict, t);
    vector<std::pair<tuple, int> >::iterator it_modulo_map;
    int index_id = 1;
    int actual_patterns_items_size = 0;
    for (it_modulo_map = patterns.begin();it_modulo_map != patterns.end();it_modulo_map++) {
        tuple t = it_modulo_map->first;
#ifdef TEST_PARAMS
        cout<<"tuple:";
        t.print();
        cout<<"----";
#endif
        //since now there is only 2 items in the tuple.
        tuple op = t.getFirstNitems(1);
        tuple op1 = t.getLastItem();
#ifdef TEST_PARAMS
        cout<<"split tuple=";
        op.print();
        cout<< "---";
        op1.print();
        cout<<endl;
#endif
        if (!isTuplePresent(new_modulo_map, op)) {
#ifdef TEST_PARAMS
            cout<<"adding tuple to api_h=";
            op.print();
            cout<<"id assigned="<<index_id<<endl;
#endif
            actual_patterns_items_size += op.size();
            new_modulo_map.push_back(std::pair<tuple, int>(op, index_id));
            index_id++;
        }
        if (!isTuplePresent(new_modulo_map, op1)) {
#ifdef TEST_PARAMS
            cout<<"adding tuple to api_h=";
            op1.print();
            cout<<"id assigned="<<index_id<<endl;
#endif
            actual_patterns_items_size += op1.size();
            new_modulo_map.push_back(std::pair<tuple, int>(op1, index_id)); 
            index_id++;
        }
    }

#ifdef TEST_PARAMS
    for (it_modulo_map = new_modulo_map.begin(); it_modulo_map != new_modulo_map.end();it_modulo_map++) {
        cout<<"id[";
        it_modulo_map->first.print();
        cout<<"]="<<it_modulo_map->second<<endl; 
    }
#endif
    //cout<<"actual_patterns_items_size:"<<actual_patterns_items_size<<endl;
    int index_items_lookup_size = 3 * new_modulo_map.size();// (index_id, start, length)
    //cout<<"index_items_lookup_size :"<<index_items_lookup_size<<endl;
    unsigned int *actual_patterns_items = (unsigned int *) malloc(actual_patterns_items_size * sizeof (unsigned int));
    unsigned int *index_items_lookup = (unsigned int *) malloc(index_items_lookup_size * sizeof (unsigned int));
    int start_offset = 0;
    int counter = 0;
    for (it_modulo_map = new_modulo_map.begin(); it_modulo_map != new_modulo_map.end();it_modulo_map++) {
        tuple t = it_modulo_map->first;
        //cout<<"makeiid tuple:";
        //t.print();
        //cout<<"--- index_id="<<it_modulo_map->second<<" start="<<start_offset<<"length="<<t.size()<<endl;
        index_items_lookup[counter] = it_modulo_map->second;
        index_items_lookup[counter+1] = start_offset; 
        index_items_lookup[counter+2] = t.size();
        //cout<<"--- index_id_tuple=("<<index_items_lookup[counter]<<","<<index_items_lookup[counter+1]<<","<<index_items_lookup[counter+2]<<endl;
        counter +=3;
        for (int i =0; i < t.size();i++) {
            actual_patterns_items[start_offset] = t.get(i);
            //cout<<"api_h["<<start_offset<<"]="<<actual_patterns_items[start_offset]<<endl;
            start_offset++;
        }
    }
#ifdef TEST_PARAMS
    for (int i = 0;i < index_items_lookup_size;i+=3) {
        cout<<"iil_h["<<i<<"]="<<"("<<index_items_lookup[i]<<","<<index_items_lookup[i+1]<<","<<index_items_lookup[i+2]<<")"<<endl;
    }
#endif
    // now create the new encoded array
    unsigned int *new_new_patterns;
    unsigned int *new_new_patterns_d;
    int new_new_patterns_size = patterns.size(); 
    new_new_patterns = (unsigned int *) malloc(new_new_patterns_size * sizeof (unsigned int));
    counter = 0;
    int mul_factor = (int)(pow(10.0, (double)power));
    for (it_modulo_map = patterns.begin();it_modulo_map != patterns.end();it_modulo_map++) {
        tuple t = it_modulo_map->first;
        //since now there is only 2 items in the tuple.
        tuple op = t.getFirstNitems(1);
        tuple op1 = t.getLastItem();
        int code1 = getTupleValue(new_modulo_map, op);
        int code2 = getTupleValue(new_modulo_map, op1);
        if (code1 == INVALID || code2 == INVALID) continue;
        
        int newcode = code1 * mul_factor + code2;
        //cout<<"add to nnp["<<counter<<"]="<<newcode<<endl;
        new_new_patterns[counter++] = newcode;
    }
   
    //may apply radix sort to sort it
    qsort(new_new_patterns, sizeof(new_new_patterns)/sizeof(new_new_patterns[0]), sizeof(new_new_patterns[0]), compare);
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    //send the array to device
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&new_new_patterns_d, new_new_patterns_size * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
    
    
    startTime(&timer);
    cuda_ret = hipMemcpy(new_new_patterns_d, new_new_patterns, new_new_patterns_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
#ifdef TEST_PARAMS
    for (int i = 0;i < new_new_patterns_size;i++) {
        cout<<"nnp["<<i<<"]="<<new_new_patterns[i]<<endl;    
    }
#endif
    //#########################################################//
    //############# start of second phase######################// 
    // calculate parameters again for second phase
    k = counter;
    cout<<"#############new k="<<k<<endl;
    maskLength = pow(float(k), 2);
    cout <<"############new maskLength ="<<maskLength<<endl;
    int *mask1_h = (int*)malloc(maskLength * sizeof(int));
    int* mask1_d;//mask matrix
    cout<<"alloc mask1 matrix"<<endl;
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&mask1_d, maskLength * sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
    
    cuda_ret = hipMemset(mask1_d, -1, maskLength * sizeof(int));
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * MAX_ITEM_PER_SM));
    grid_dim.y = 1;
    grid_dim.z = 1;
    //#########################################################//
    //#############selfJoinKernel##############################// 
    cout<<"self join launched with <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    selfJoinKernel<<<grid_dim, block_dim>>>(new_new_patterns_d, mask1_d, k, power);
    hipDeviceSynchronize();
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
    //TBD:remove. only for test
    cout<<"copy mask back to host"<<endl;
    startTime(&timer);
    hipMemcpy(mask1_h, mask1_d, maskLength * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
#ifdef TEST_PARAMS
    /*cout<<"################mask_h after join#############"<<endl;
    for (int i = 0;i < k; i++) {
        for (int j = 0;j < k;j++) {
            cout<<mask1_h[i*k + j]<<" ";
        }
        cout<<endl;
    }*/
#endif
    unsigned int *actual_patterns_items_d;
    unsigned int *index_items_lookup_d;
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&actual_patterns_items_d, actual_patterns_items_size * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&index_items_lookup_d, index_items_lookup_size * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);

    startTime(&timer);
    cuda_ret = hipMemcpy(actual_patterns_items_d, actual_patterns_items, actual_patterns_items_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy input to the device");
    cuda_ret = hipMemcpy(index_items_lookup_d, index_items_lookup, index_items_lookup_size * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy input to the device");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);

    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil((num_transactions) / (1.0 * MAX_TRANSACTION_PER_SM));
    grid_dim.y = 1;
    grid_dim.z = 1;
    //##############################################################################
    //"################findHigherPatternFrequencyGPU###############################
    cout<<"findHigherPatternFrequencyGPU launched with <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    findHigherPatternFrequencyGPU<<<grid_dim, block_dim>>>(d_input, d_offsets,
                                  num_transactions, 
                                  num_elements, new_new_patterns_d,
                                  mask1_d, k, actual_patterns_items_d,
                                  index_items_lookup_d, power,
                                  actual_patterns_items_size,
                                  index_items_lookup_size, maskLength);
    hipDeviceSynchronize();
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);

    // prune the matrix
    block_dim.x = BLOCK_SIZE;
    block_dim.y = BLOCK_SIZE;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = (int) ceil(k / (1.0 * block_dim.y));
    grid_dim.z = 1;
    //######################################################################
    //"################pruneMultipleGPU_kernel##############################
    cout<<"pruneMultipleGPU <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    pruneMultipleGPU_kernel<<<grid_dim, block_dim>>>(mask1_d, k, MIN_SUPPORT);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch pruneMultipleGPU");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);

#ifdef TEST_PARAMS
    cout<<"copy mask back to host"<<endl;
    startTime(&timer);
    hipMemcpy(mask1_h, mask1_d, maskLength * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    cout<<"################mask_h after join#############"<<endl;
    for (int i = 0;i < k; i++) {
        for (int j = 0;j < k;j++) {
            cout<<mask1_h[i*k + j]<<"   ";
        }
        cout<<endl;
    }
#endif
    unsigned int *ci1_dn;
    unsigned int *ci1_hn;
    ci1_hn = (unsigned int*) malloc(k * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&ci1_dn, k * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(ci1_dn, 0, k * sizeof(unsigned int));
    //find combinations available
    block_dim.x = BLOCK_SIZE;
    block_dim.y = BLOCK_SIZE;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = (int) ceil(k / (1.0 * block_dim.y));
    grid_dim.z = 1;

    //######################################################################
    //"################CombinationsAvailable_kernel##############################
    cout<<"combinationsAvailable_kernel <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    combinationsAvailable_kernel<<<grid_dim, block_dim>>>(mask1_d, ci1_dn, k, maskLength);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch combinationsAvailable_kernel");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
    
    startTime(&timer);
    hipMemcpy(ci1_hn, ci1_dn, k * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
#ifdef TEST_PARAMS
    for (int i = 0; i < k; i++) {
        cout<<"ci1_dn["<<i<<"]="<<ci1_hn[i]<<endl;    
    }
#endif
    unsigned int *ci1_hnx;
    unsigned int *ci1_dnx;
    startTime(&timer);
    ci1_hnx = (unsigned int*) malloc(k * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&ci1_dnx, k * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(ci1_dnx, 0, k * sizeof(unsigned int));
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
    
    
    // prescan it to get offsets
    startTime(&timer);
    preScan(ci1_dnx, ci1_dn, k);
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    hipMemcpy(ci1_hnx, ci1_dnx, k * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");    
#ifdef TEST_PARAMS
    cout<<"scan op"<<endl;
    for (int i = 0; i < k; i++) {
        cout<<"ci_dnx["<<i<<"]="<<ci1_hnx[i]<<endl;    
    }
#endif
    unsigned int *sparseM_h1;
    unsigned int *sparseM_d1;
    unsigned int sparse_matrix_size1 = ci1_hnx[k-1];
    cout<<"allocating sparse matrix for size"<<sparse_matrix_size1<<endl; 
    startTime(&timer);
    sparseM_h1 = (unsigned int*) malloc(3 * sparse_matrix_size1 * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&sparseM_d1, 3 * sparse_matrix_size1 * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(sparseM_d1, 0, 3 * sparse_matrix_size1 * sizeof(unsigned int));
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalAllocTime += elapsedTime(timer);
     
    
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = 1;
    grid_dim.z = 1;
    cout<<" convert2Sparse kernel <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    convert2Sparse<<<grid_dim, block_dim>>>(mask1_d, ci1_dnx, sparseM_d1, sparse_matrix_size1, k);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch convert2Sparse");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
    startTime(&timer);
    hipMemcpy(sparseM_h1, sparseM_d1, 3 * sparse_matrix_size1 * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    
#ifdef TEST_PARAMS
    cout<<"sparse op1(row,col,val)"<<endl;
    for (int i = 0; i < sparse_matrix_size1; i++) {
        cout<<"sparse("<<sparseM_h1[i]<<","<<sparseM_h1[i + sparse_matrix_size1]<<","<<sparseM_h1[i + 2*sparse_matrix_size1]<<")"<<endl;    
    }
#endif
    cout<<"Making vector maps2"<<endl;
    startTime(&timer);
    vector<std::pair<tuple, int> > patterns1;
    for (int i = 0; i< sparse_matrix_size1;i++) {
        tuple t(new_new_patterns[sparseM_h1[i]], new_new_patterns[sparseM_h1[i + sparse_matrix_size1]]);
        int item = sparseM_h1[i + 2 * sparse_matrix_size1];
        patterns1.push_back(std::pair<tuple, unsigned int>(t, item));    
    }
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    totalRunTime += elapsedTime(timer);
    cout<<"map size"<<patterns1.size()<<endl;
#ifdef TEST_PARAMS
    for (it = patterns1.begin(); it != patterns1.end();it++) {
        it->first.print();
        cout<<"="<<it->second<<endl;
    }
#endif
    cout<<"TotalRunTime="<<totalRunTime<<endl;
    cout<<"TotalAllocTime="<<totalAllocTime<<endl;
exit:
    if (trans_offset) {
        free(trans_offset);
    }
    if (transactions) {
        free(transactions);
    }
    if (ci_h) {
        free(ci_h);
    }
    if (li_h) {
        free(li_h);    
    }
    if (mask_h) {
        free(mask_h);    
    }
    if (ci_hn) {
        free(ci_hn);    
    }
    if (ci_hnx) {
        free(ci_hnx);    
    }
    if (sparseM_h) {
        free(sparseM_h);
    }
    if(actual_patterns_items) {
        free(actual_patterns_items);
    }
    if (index_items_lookup) {
        free(index_items_lookup);
    }
    if (new_new_patterns) {
        free(new_new_patterns);
    }
    if (mask1_h) {
        free(mask1_h);
    }
    if (ci1_hn) {
        free(ci1_hn);    
    }
    if (ci1_hnx) {
        free(ci1_hnx);    
    } 
    if (sparseM_h1) {
        free(sparseM_h1);
    }
    hipFree(d_offsets);
    hipFree(d_input);
    hipFree(ci_d);
    hipFree(li_d);
    hipFree(mask_d);
    hipFree(ci_dn);
    hipFree(ci_dnx);
    hipFree(sparseM_d);
    hipFree(new_new_patterns_d);
    hipFree(mask1_d);
    hipFree(ci1_dn); 
    hipFree(ci1_dnx); 
    hipFree(sparseM_d1);
    hipFree(actual_patterns_items_d);
    hipFree(index_items_lookup_d);
    cout<<"program end";

}

bool pair_compare(const pair<short unsigned int, unsigned int>& p1,const pair<short unsigned int, unsigned int>& p2) {
    return p1.second < p2.second;    
}
