#include "hip/hip_runtime.h"
//#include <stdio.h>
#include <iostream>
#include<algorithm>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>
#include <fstream>
#include "defs.h"
#include "support.h"
#include "kernel.cu"
#include "kernel_prescan.cu"
#include<vector>
#include<utility>
using namespace std;


bool pair_compare(const pair<short unsigned int, unsigned int>& p1,const pair<short unsigned int, unsigned int>& p2);
int main(int argc, char* argv[])
{
    char *line = NULL;
    size_t len = 0;
    unsigned int lines = 0;
    unsigned int count = 0;
    char *ln, *nptr;

    unsigned int *transactions = NULL;
    unsigned int *trans_offset = NULL;
    unsigned int *ci_h = NULL;//bins array for histogram op
    //unsigned int *flist = NULL;
    //unsigned short *flist_key_16_index = NULL;

    unsigned int element_id = 0;

    transactions = (unsigned int *) malloc(MAX_NUM_ELEMENTS * sizeof(unsigned int));
    trans_offset = (unsigned int *) malloc((MAX_NUM_ELEMENTS + 1) * sizeof(unsigned int));
    ci_h = (unsigned int *) malloc(MAX_UNIQUE_ITEMS * sizeof(unsigned int));
    //flist = (unsigned int *) malloc(max_unique_items * sizeof(unsigned int));
    //flist_key_16_index = (unsigned short*) malloc(max_unique_items * sizeof(unsigned short));

//    memset(flist_key_16_index, 0xFFFF, max_unique_items * sizeof(unsigned short));

    lines = 0;
    element_id = 0;
    ifstream fp1("topic-3.txt");
    string curline, space(" ");
    if (fp1.is_open()) {
        cout<<"file opened"<<endl;
        trans_offset[0] = 0;
        while(getline(fp1, curline) && lines < MAX_TRANSACTIONS) {
            count = 0;
            istringstream s(curline); 
            string st;
            while(getline(s, st, ' ') && count < MAX_ITEM_PER_TRANSACTION) {
                int item = atol(st.c_str());
                if (item < MAX_UNIQUE_ITEMS) {
                    // add an item only if it is in the range [0,max_unique_items)
                    transactions[element_id++] = atol(st.c_str());
                    count++;
                }
            }
            if (count > 0) {
                // consider this transaction if there is atleast one item
                trans_offset[lines + 1] = trans_offset[lines] + count;
                lines++;
            }
        }
    } else {
        cout<<"error in reading from file"<<endl;
        return 0;
    }
    fp1.close();
    unsigned int num_elements = element_id;
    unsigned int num_transactions = lines;
    cout<<"Number of Transactions = "<<num_transactions<<endl;
    cout<<"num_elements in transactions array = "<<num_elements<<endl;
    #ifdef TEST_PARAMS
    /*for (int i = 0; i < num_elements; i++){
        cout<<transactions[i]<<" ";
    }
    cout<<endl;
    for (int i = 0; i <= num_transactions; i++) {
       cout<<"(i,offset)"<<i<<","<<trans_offset[i]; 
    }*/
    #endif

    //calculate max power
    int power = 1;
    while ((MAX_UNIQUE_ITEMS / (int)(pow(10.0, (double)power))) != 0) {
        power += 1;
    }
    cout<<"max power = "<<power<<endl;

    //check for max item exceed
    if (num_elements > MAX_NUM_ELEMENTS) {
        cout<<"Error: Elements exceeding NUM_ELEMENTS. Exiting...";
        return -1;
    }
    /////////////////////////////////////////////////////////////////////////////////////
    /////////////////////// Device Variables Initializations ///////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////
    unsigned int *d_input;//
    unsigned int *d_offsets;
    unsigned int *ci_d;//bins array - each index corrosponds to an item
    hipDeviceProp_t deviceProp;
    Timer timer;
    hipError_t cuda_ret;
    hipGetDeviceProperties(&deviceProp, 0);
    cout<<"Allocating device variables...";
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&d_input, num_elements * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&d_offsets, (num_transactions + 1) * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    cuda_ret = hipMalloc((void**)&ci_d, MAX_UNIQUE_ITEMS * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMemset(ci_d, 0, MAX_UNIQUE_ITEMS * sizeof(unsigned int));
    cuda_ret = hipMemcpy(d_input, transactions, num_elements * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy input to the device");
    cuda_ret = hipMemcpy(d_offsets, trans_offset, (num_transactions+1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy input to the device");
    
    dim3 grid_dim, block_dim;
    block_dim.x = BLOCK_SIZE; 
    block_dim.y = 1; block_dim.z = 1;
    grid_dim.x = ceil(num_elements / (1.0 * BLOCK_SIZE)); 
    grid_dim.y = 1; grid_dim.z = 1;
    cout<<"launching histogram kernel(grid, block):"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    histogram_kernel<<<grid_dim, block_dim, MAX_UNIQUE_ITEMS * sizeof(unsigned int)>>>(d_input, ci_d, num_elements);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch Histogram kernel");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    // prune the histogram op 
    block_dim.x = BLOCK_SIZE; 
    block_dim.y = 1; block_dim.z = 1;
    grid_dim.x = ceil(MAX_UNIQUE_ITEMS / (1.0 * BLOCK_SIZE)); 
    grid_dim.y = 1; grid_dim.z = 1;
    cout<<"launching pruning kernel(grid, block):"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    pruneGPU_kernel<<<grid_dim, block_dim>>>(ci_d, MAX_UNIQUE_ITEMS, MIN_SUPPORT);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    
    cout<<"copying hist op back to host"<<endl;
    startTime(&timer);
    hipMemcpy(ci_h, ci_d, MAX_UNIQUE_ITEMS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
#ifdef TEST_PARAMS
    /*cout<<"histogram output after pruning:"<<endl;
    for (int i = 0; i < MAX_UNIQUE_ITEMS; i++) {
        cout<<"ci_h["<<i<<"]="<<ci_h[i]<<endl;   
    } */    
#endif
    unsigned int *li_h; // this list contains the actual items which passed min support test
    unsigned int  k = 0; //count of actual items which passed min support test
    for (int i = 0;i<MAX_UNIQUE_ITEMS;i++) {
        if (ci_h[i] != 0) {
            k++;    
        }    
    }
    cout<<"num items with good support count="<<k<<endl;
    li_h = (unsigned int *) malloc(k * sizeof(unsigned int));
    /*if (li_h ==  NULL) {
        cout<<"faild to alloc li_h...exiting!"<<endl;
        goto exit;    
    }*/
    int li_count = 0;
    for (int i = 0;i<MAX_UNIQUE_ITEMS;i++) {
        if (ci_h[i] != 0) {
            li_h[li_count++] = i; 
        } 
    }

//#ifdef TEST_PARAMS
    cout<<"li_h after pruning:"<<endl;
    for (int i = 0; i < k; i++) {
        cout<<"li_h["<<i<<"]="<<li_h[i]<<endl;   
    }
//#endif
    unsigned int *li_d;
    cuda_ret = hipMalloc((void**)&li_d, k * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemcpy(li_d, li_h, k * sizeof(unsigned int), hipMemcpyHostToDevice);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy li_h to device");
    
    int maskLength = pow(float(k), 2);
    cout <<"maskLength ="<<maskLength<<endl;
    int *mask_h = (int*)malloc(maskLength * sizeof(int));
    int* mask_d;//mask matrix
    cout<<"alloc mask matrix"<<endl;
    startTime(&timer);
    cuda_ret = hipMalloc((void**)&mask_d, maskLength * sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil((maskLength) / (1.0 * block_dim.x));
    grid_dim.y = 1;
    grid_dim.z = 1;
    cout<<"init mask"<<endl;
    startTime(&timer);
    initializeMaskArray<<<grid_dim, block_dim>>>(mask_d, maskLength);
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    hipDeviceSynchronize();

    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * MAX_ITEM_PER_SM));
    grid_dim.y = 1;
    grid_dim.z = 1;
    cout<<"self join launched with <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    selfJoinKernel<<<grid_dim, block_dim>>>(li_d, mask_d, k, power);
    hipDeviceSynchronize();
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
  
    // TBD:to test. remove in final code
    cout<<"copy mask back to host"<<endl;
    startTime(&timer);
    hipMemcpy(mask_h, mask_d, maskLength * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
#ifdef TEST_PARAMS
    cout<<"################mask_h after join#############"<<endl;
    for (int i = 0;i < maskLength; i++) {
        cout<<"mask["<<i<<"]="<<mask_h[i]<<endl;   
        
    }
#endif
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil((num_transactions) / (1.0 * MAX_TRANSACTION_PER_SM));
    grid_dim.y = 1;
    grid_dim.z = 1;
    cout<<"findFrequencyGPU <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    findFrequencyGPU_kernel<<<grid_dim, block_dim>>>(d_input, d_offsets, num_transactions, num_elements, li_d, mask_d, k, maskLength);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch findFrequencyGPU_kernel");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    //prune the 2d mask matrix
    block_dim.x = BLOCK_SIZE;
    block_dim.y = BLOCK_SIZE;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = (int) ceil(k / (1.0 * block_dim.y));
    grid_dim.z = 1;
    //cout<<"gridy"<<grid_dim.y<<endl;
    cout<<"pruneMultipleGPU <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    pruneMultipleGPU_kernel<<<grid_dim, block_dim>>>(mask_d, k, MIN_SUPPORT);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch pruneMultipleGPU");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;

    cout<<"copy mask back to host"<<endl;
    startTime(&timer);
    hipMemcpy(mask_h, mask_d, maskLength * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
#ifdef TEST_PARAMS
    cout<<"################mask_h after findFrequencyGPU_kernel and Prune#############"<<endl;
    for (int i = 0;i < maskLength; i++) {
        cout<<"mask["<<i<<"]="<<mask_h[i]<<endl;   
        
    }
#endif
    //now we need to convert the mask array to a sparse matrix in parallel
    // this means we need to find number of non zero entries in each row of mask matrix
    // and the allocate memory equal to total number of non zero items.
    // each thread can then directly work on an offset into the array, 
    // obtained by perorming a exclusive scan.
    unsigned int *ci_dn;
    unsigned int *ci_hn;
    ci_hn = (unsigned int*) malloc(k * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&ci_dn, k * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(ci_dn, 0, k * sizeof(unsigned int));
    
    block_dim.x = BLOCK_SIZE;
    block_dim.y = BLOCK_SIZE;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = (int) ceil(k / (1.0 * block_dim.y));
    grid_dim.z = 1;

    cout<<"combinationsAvailable_kernel <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    combinationsAvailable_kernel<<<grid_dim, block_dim>>>(mask_d, ci_dn, k, maskLength);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch combinationsAvailable_kernel");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    hipMemcpy(ci_hn, ci_dn, k * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
//#ifdef TEST_PARAMS
    for (int i = 0; i < k; i++) {
        cout<<"ci_dn["<<i<<"]="<<ci_hn[i]<<endl;    
    }
//#endif
    // prescan the ci_hn array
    unsigned int *ci_hnx;
    unsigned int *ci_dnx;
    ci_hnx = (unsigned int*) malloc(k * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&ci_dnx, k * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(ci_dnx, 0, k * sizeof(unsigned int));

    preScan(ci_dnx, ci_dn, k);
    hipMemcpy(ci_hnx, ci_dnx, k * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
//#ifdef TEST_PARAMS
    cout<<"scan op"<<endl;
    for (int i = 0; i < k; i++) {
        cout<<"ci_dnx["<<i<<"]="<<ci_hnx[i]<<endl;    
    }
//#endif

    unsigned int *sparseM_h;
    unsigned int *sparseM_d;
    unsigned int sparse_matrix_size = ci_hnx[k-1];
    cout<<"allocating sparse matrix for size"<<sparse_matrix_size<<endl; 
    sparseM_h = (unsigned int*) malloc(3 *sparse_matrix_size * sizeof (unsigned int));
    cuda_ret = hipMalloc((void**)&sparseM_d, 3 * sparse_matrix_size * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(sparseM_d, 0, 3 * sparse_matrix_size * sizeof(unsigned int));
    block_dim.x = BLOCK_SIZE;
    block_dim.y = 1;
    block_dim.y = 1;
    grid_dim.x = (int) ceil(k / (1.0 * block_dim.x));
    grid_dim.y = 1;
    grid_dim.z = 1;
    cout<<" convert2Sparse kernel <grid,block>"<<grid_dim.x<<","<<block_dim.x<<endl;
    startTime(&timer);
    convert2Sparse<<<grid_dim, block_dim>>>(mask_d, ci_dnx, sparseM_d, sparse_matrix_size, k);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch convert2Sparse");
    stopTime(&timer); cout<<elapsedTime(timer)<<endl;
    
    hipMemcpy(sparseM_h, sparseM_d, 3 * sparse_matrix_size * sizeof(int), hipMemcpyDeviceToHost);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to copy histogram op back to host");
#ifdef TEST_PARAMS
    cout<<"sparse op(row,col,val)"<<endl;
    for (int i = 0; i < sparse_matrix_size; i++) {
        cout<<"sparse("<<sparseM_h[i]<<","<<sparseM_h[i + sparse_matrix_size]<<","<<sparseM_h[i + 2*sparse_matrix_size]<<")"<<endl;    
    }
#endif
exit:
    if (trans_offset) {
        free(trans_offset);
    }
    if (transactions) {
        free(transactions);
    }
    if (ci_h) {
        free(ci_h);
    }
    if (li_h) {
        free(li_h);    
    }
    if (mask_h) {
        free(mask_h);    
    }
    if (ci_hn) {
        free(ci_hn);    
    }
    if (ci_hnx) {
        free(ci_hnx);    
    }
    hipFree(d_offsets);
    hipFree(d_input);
    hipFree(ci_d);
    hipFree(li_d);
    hipFree(mask_d);
    hipFree(ci_dn);
    hipFree(ci_dnx);
    cout<<"program end";

}

bool pair_compare(const pair<short unsigned int, unsigned int>& p1,const pair<short unsigned int, unsigned int>& p2) {
    return p1.second < p2.second;    
}
